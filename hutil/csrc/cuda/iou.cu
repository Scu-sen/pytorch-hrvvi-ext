
#include <hip/hip_runtime.h>
// #include <torch/extension.h>

// template <typename T> __device__ inline float iou_11(const T *a, const T *b)
// {
//     T left = max(a[0], b[0]), right = min(a[2], b[2]);
//     T top = max(a[1], b[1]), bottom = min(a[3], b[3]);
//     T width = max(right - left, (T)0), height = max(bottom - top, (T)0);
//     T interS = width * height;
//     T Sa = (a[2] - a[0]) * (a[3] - a[1]);
//     T Sb = (b[2] - b[0]) * (b[3] - b[1]);
//     return interS / (Sa + Sb - interS);
// }

// template <typename T>
// __global__ void iou_nm_forward(const T *boxes1, const T *boxes2, const int n,
//                                const int m, T *ious) {
//     int i = blockIdx.x * blockDim.x + threadIdx.x;
//     int j = blockIdx.y * blockDim.y + threadIdx.y;

//     if (i >= n || j >= m)
//         return;

//     T iou = iou_11(boxes1 + i * 4, boxes2 + j * 4);
//     ious[i * m + j] = iou;
// }

// std::tuple<at::Tensor, at::Tensor>
// iou_nm_forward_cuda(const at::Tensor &boxes1, const at::Tensor &boxes2) {
//     AT_ASSERTM(boxes1.device().is_cuda(), "boxes1 must be a CUDA tensor");
//     AT_ASSERTM(boxes2.device().is_cuda(), "boxes2 must be a CUDA tensor");

//     at::TensorArg boxes1_t{boxes1, "boxes1", 1}, boxes2_t{boxes2, "boxes2",
//     2};

//     at::CheckedFrom c = "iou_nm_forward_cuda";
//     at::checkAllSameGPU(c, {boxes1_t, boxes2_t});
//     at::checkAllSameType(c, {boxes1_t, boxes2_t});

//     at::cuda::CUDAGuard device_guard(boxes1.device());

//     auto n = boxes1.size(0);
//     auto m = boxes2.size(0);

//     at::Tensor ious = at::zeros({n, m}, boxes1.options());

//     cudaStream_t stream = at::cuda::getCurrentCUDAStream();

//     const dim3 blockSize(32, 32);
//     const dim3 numBlocks(THCCeilDiv(n, 32L), THCCeilDiv(m, 32L));

//     if (ious.numel() == 0) {
//         THCudaCheck(cudaGetLastError());
//         return ious;
//     }

//     iou_nm_forward<<<numBlocks, blockSize>>>(boxes1, boxes2, n, m, ious);

//     AT_DISPATCH_FLOATING_TYPES_AND_HALF(
//         boxes1.type(), "iou_nm_forward_cuda", [&] {
//             iou_nm_forward<scalar_t><<<numBlocks, blockSize, 0, stream>>>(
//                 boxes1.contiguous().data<scalar_t>(),
//                 boxes2.contiguous().data<scalar_t>(), n, m,
//                 ious.contiguous().data<scalar_t>());
//         });
//     THCudaCheck(cudaGetLastError());
//     return ious
// }

// template <typename T>
// __device__ inline void iou_11_backward(T *dbox1, T *dbox2, const T dout,
//                                        const T *box1, const T *box2,
//                                        const T out) {
//     if (out == 0) {
//         return;
//     }

//     T ix1 = box1[0];
//     T iy1 = box1[1];
//     T ix2 = box1[2];
//     T iy2 = box1[3];
//     T iw = ix2 - ix1;
//     T ih = iy2 - iy1;
//     T iarea = iw * ih;

//     T jx1 = box2[0];
//     T jy1 = box2[1];
//     T jx2 = box2[2];
//     T jy2 = box2[3];
//     T jw = jx2 - jx1;
//     T jh = jy2 - jy1;
//     T jarea = jw * jh;

//     T xx1 = std::max(ix1, jx1);
//     T yy1 = std::max(iy1, jy1);
//     T xx2 = std::min(ix2, jx2);
//     T yy2 = std::min(iy2, jy2);
//     T w = std::max(static_cast<T>(0.0), xx2 - xx1);
//     T h = std::max(static_cast<T>(0.0), yy2 - yy1);
//     T inter_area = w * h;
//     T union_area = iarea + jarea - inter_area;

//     T darea = dout * inter_area / (union_area * union_area);

//     atomicAdd(dbox1, ih * darea);
//     atomicAdd(dbox1 + 1, iw * darea);
//     atomicAdd(dbox1 + 2, -ih * darea);
//     atomicAdd(dbox1 + 3, -iw * darea);

//     atomicAdd(dbox2, jh * darea);
//     atomicAdd(dbox2 + 1, jw * darea);
//     atomicAdd(dbox2 + 2, -jh * darea);
//     atomicAdd(dbox2 + 3, -jw * darea);

//     T dinter = dout * (inter_area + union_area) / (union_area * union_area);
//     T dw = h * dinter;
//     T dh = w * dinter;

//     if (ix1 >= jx1) {
//         atomicAdd(dbox1, -dw);
//     } else {
//         atomicAdd(dbox2, -dw);
//     }

//     if (iy1 >= jy1) {
//         atomicAdd(dbox1 + 1, -dh);
//     } else {
//         atomicAdd(dbox2 + 1, -dh);
//     }

//     if (ix2 <= jx2) {
//         atomicAdd(dbox1 + 2, dw);
//     } else {
//         atomicAdd(dbox2 + 2, dw);
//     }

//     if (iy2 <= jy2) {
//         atomicAdd(dbox1 + 3, dh);
//     } else {
//         atomicAdd(dbox2 + 3, dh);
//     }
// }

// template <typename T>
// __global__ void iou_nm_backward(T *dboxes1, T *dboxes2, const T *dout,
//                                 const T *boxes1, const T *boxes2, const int
//                                 n, const int m, const T *ious) {
//     int i = blockIdx.x * blockDim.x + threadIdx.x;
//     int j = blockIdx.y * blockDim.y + threadIdx.y;

//     if (i >= n || j >= m)
//         return;

//     iou_11_backward(dboxes1 + i * 4, dboxes2 + j * 4, dout[i * m + j],
//                     boxes1 + i * 4, boxes2 + j * 4, ious[i * m + j]);
// }

// at::Tensor iou_nm_backward_cuda(const at::Tensor &dout,
//                                 const at::Tensor &boxes1,
//                                 const at::Tensor &boxes2,
//                                 const at::Tensor &ious) {
//     // Check if input tensors are CUDA tensors
//     AT_ASSERTM(dout.device().is_cuda(), "dout must be a CUDA tensor");
//     AT_ASSERTM(boxes1.device().is_cuda(), "boxes1 must be a CUDA tensor");
//     AT_ASSERTM(boxes2.device().is_cuda(), "boxes2 must be a CUDA tensor");
//     AT_ASSERTM(ious.device().is_cuda(), "ious must be a CUDA tensor");

//     at::TensorArg dout_t{dout, "dout", 1}, boxes1_t{boxes1, "boxes1", 2},
//         boxes2_t{boxes2, "boxes2", 3}, ious_t{ious, "ious", 4};

//     at::CheckedFrom c = "iou_nm_backward_cuda";
//     at::checkAllSameGPU(c, {dout_t, boxes1_t, boxes2_t, ious_t});
//     at::checkAllSameType(c, {dout_t, boxes1_t, boxes2_t, ious_t});

//     at::cuda::CUDAGuard device_guard(dout.device());

//     auto n = boxes1.size(0);
//     auto m = boxes2.size(0);

//     at::Tensor dboxes1 = at::zeros({n, 4}, boxes1.options());
//     at::Tensor dboxes2 = at::zeros({m, 4}, boxes2.options());

//     cudaStream_t stream = at::cuda::getCurrentCUDAStream();

//     const dim3 blockSize(32, 32);
//     const dim3 numBlocks(THCCeilDiv(n, 32L), THCCeilDiv(m, 32L));

//     if (dout.numel() == 0) {
//         THCudaCheck(cudaGetLastError());
//         return std::make_tuple(dboxes1, dboxes2);
//     }

//     int n_stride = dout.stride(0);
//     int m_stride = dout.stride(1);

//     AT_DISPATCH_FLOATING_TYPES_AND_HALF(
//         dout.type(), "iou_nm_backward_cuda", [&] {
//             iou_nm_backward<scalar_t><<<numBlocks, blockSize, 0, stream>>>(
//                 dboxes1.data<scalar_t>(), dboxes2.data<scalar_t>(),
//                 dout.contiguous().data<scalar_t>(),
//                 boxes1.contiguous().data<scalar_t>(),
//                 boxes2.contiguous().data<scalar_t>(),
//                 ious.contiguous().data<scalar_t>());
//         });
//     THCudaCheck(cudaGetLastError());
//     return std::make_tuple(dboxes1, dboxes2);
// }